#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    if (nDevices == 0) {
        printf("No CUDA devices found.\n");
        return -1;
    }

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
    }

    return 0;
}