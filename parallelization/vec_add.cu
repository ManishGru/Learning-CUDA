#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        out[tid] = a[tid] + b[tid];
}

int main()
{
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    hipMalloc((void **)&d_a, sizeof(float) * N);
    hipMalloc((void **)&d_b, sizeof(float) * N);
    hipMalloc((void **)&d_out, sizeof(float) * N);

    for (int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    int threadNum = 256;
    int blockNum = (N + threadNum) / threadNum;
    vector_add<<<blockNum, threadNum>>>(d_out, d_a, d_b, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for (int i = 0; i < N; i++)
    {
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
    return 0;
}